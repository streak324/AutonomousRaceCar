#include "hip/hip_runtime.h"
#include <stdio.h>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "zedMagic.h"
#define square(x) x*x
#define THRESHOLD 70
#define THRESHOLD2 12000

#define AREA H_O*W_O
#define CROPPED_AREA W_I*H_O*3*2

__global__
void saxpy(int n, float a, float *x, float *y) {
    int i=blockIdx.x*blockDim.x + threadIdx.x;
    if (i<n) y[i]=a*x[i]+y[i];
}

__global__
void edgeMath(unsigned char* bw, float* edges) {
    int index = blockIdx.x*blockDim.x + threadIdx.x;
    int i = index/W_O;
    int j = index%W_O;
    if (i==0 || i==H_O-1) edges[index] = 0;
    else {
        if (j==0 || j==W_O-1) edges[index] = 0;
        else {
            int dx,dy;
            int newW = W_O;
                dx = bw[index+newW+1] + 2*bw[index+1] + bw[index-newW+1] \
                - (bw[index+newW-1] + 2*bw[index-1] + bw[index-newW-1]);
                dy = bw[index-newW-1] + 2*bw[index-newW] + bw[index-newW+1]\
                - (bw[index+newW-1] + 2*bw[index+newW] + bw[index+newW+1]);
                double sgm = (square(dx) + square(dy));
            edges[index] = sgm;
        }
    }
}

__global__
void edgeMath(unsigned char* bw, unsigned int* edges) {
    int index = blockIdx.x*blockDim.x + threadIdx.x;
    int i = index/W_O;
    int j = index%W_O;
    if (i==0 || i==H_O-1) edges[index] = 0;
    else {
        if (j==0 || j==W_O-1) edges[index] = 0;
        else {
            int dx,dy;
            int newW = W_O;
            dx = bw[index+newW+1] + 2*bw[index+1] + bw[index-newW+1] \
                - (bw[index+newW-1] + 2*bw[index-1] + bw[index-newW-1]);
            dy = bw[index-newW-1] + 2*bw[index-newW] + bw[index-newW+1]\
                - (bw[index+newW-1] + 2*bw[index+newW] + bw[index+newW+1]);
            unsigned int sgm = dx*dx + dy*dy;
            edges[index] = sgm;
        }
    }
}

__global__
void draw_math(float theta, int roh, unsigned char *edge_data) {
    int index = blockIdx.x*blockDim.x + threadIdx.x;
    int i = index/W_O;
    int j = index%W_O;
    int x = j;
    //int y = H_O-1-i;
    int y=i;
                
    if (abs(x*cos(theta) + y*sin(theta) - roh) < 2) {
        edge_data[index] = 255;
    }
}    

/* Assume theta in radians */
void drawMath(float theta, int roh, unsigned char* edge_data) {
    draw_math<<<W_O,H_O>>>(theta,roh,edge_data);
}

int testMain(void) {
    int N=30000;//1<<20;
    float *x, *y, *d_x, *d_y;
    
    x=(float*)malloc(N*sizeof(float));
    y=(float*)malloc(N*sizeof(float));

    hipMalloc(&d_x, N*sizeof(float));
    hipMalloc(&d_y, N*sizeof(float));
    
    for (int i=0;i<N;i++) {
        x[i]=1.0f;
        y[i]=2.0f;  
    }

    hipMemcpy(d_x,x,N*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(d_y,y,N*sizeof(float),hipMemcpyHostToDevice);


    saxpy<<<(N+255)/256, 256>>>(N,2.0f,d_x,d_y);

    hipMemcpy(y,d_y,N*sizeof(float),hipMemcpyDeviceToHost);

    float maxError=0.0f;
    for (int i=0;i<N;i++) 
        maxError=max(maxError,abs(y[i]-4.0f));

    hipFree(d_x);
    hipFree(d_y);
    free(x);
    free(y);
    return 0;
}

std::vector<unsigned char> processImage(std::vector<unsigned char> imgData, int func_id) {
    switch(func_id) {
        case CUDA_BASIC: return processImageCuda(imgData);
        case CUDA_SMART: return processImageCudaExpand(imgData);
        case DUMB: return processImageDumb(imgData);
        case DUMB_MIN: return processImageDumb(imgData);
    } return processImageCuda(imgData);
} 


std::vector<unsigned char> processImageCuda(std::vector<unsigned char> imgData) {
    // Steps:
    // Read from std::vector format
    // Crop and scale
    // Grayscale    
    // Edge/SGM
    // Black and white
    // Return to std::vector format

    //Reading into an array AND cropping AND scaling at once
    unsigned char *scCrop,*d_scCrop;
    float  *edge,*d_edge;
    std::vector<unsigned char> output; 

    scCrop = (unsigned char*) malloc(AREA*sizeof(unsigned char));
    edge = (float*) malloc(AREA*sizeof(float));
    hipMalloc(&d_scCrop,AREA*sizeof(unsigned char));
    hipMalloc(&d_edge,AREA*sizeof(float));
    
    int i,j;
    int W = W_I, H = H_I;

    if (imgData.size() != W*H*3) {
        printf("ERROR dimensions wahwahwah\n");
    } 

    for (i=0;i<H_O;i++) {
        for (j=0;j<W_O;j++) {
            long sum = 0;
            for (int k = 0; k<6; k++) {
                sum += imgData[6*j + k + 6*W*i];
                sum += imgData[6*j + k + 6*W*i + 3*W];
            }
            scCrop[i*W_O + j] = sum/12;
        }
    } //scaled and cropped into array

    hipMemcpy(d_scCrop,scCrop,AREA*sizeof(unsigned char),hipMemcpyHostToDevice);
    edgeMath<<<W_O,H_O>>>(d_scCrop,d_edge);
    hipMemcpy(edge,d_edge,AREA*sizeof(float),hipMemcpyDeviceToHost);

    double max;
    for (i=0;i<H_O;i++) for (j=0;j<W_O;j++) max = (sqrt(edge[i*W_O + j])>max)? sqrt(edge[i*W_O + j]) : max;
 
    for (i=0;i<H_O;i++) for (j=0;j<W_O;j++) output.push_back( (sqrt(edge[i*W_O + j])/max*255 > THRESHOLD)? 255: 0);


    free(scCrop);
    hipFree(d_scCrop);
    free(edge);
    hipFree(d_edge);

    return output;

}

__global__
void scaleCuda(unsigned char* imgData, unsigned char* bw) {
    int index = blockIdx.x*blockDim.x + threadIdx.x;
    int i = index/W_O;
    int j = index%(W_O);
    long sum = 0;
    for (int k = 0;k<6; k++) {
        sum += imgData[6*j + k + 6*W_I*i];
        sum += imgData[6*j + k + 6*W_I*i + 3*W_I];
    }
    bw[index] = sum/12;
}

std::vector<unsigned char> processImageCudaExpand(std::vector<unsigned char> imgData) {
    unsigned char *raw, *d_raw;
    unsigned char *scCrop,*d_scCrop;
    unsigned int *edge,*d_edge;
    std::vector<unsigned char> output; 

    raw = (unsigned char*) malloc(CROPPED_AREA*sizeof(unsigned char));
    scCrop = (unsigned char*) malloc(AREA*sizeof(unsigned char));
    hipMalloc(&d_raw,CROPPED_AREA*sizeof(unsigned char));
    hipMalloc(&d_scCrop,AREA*sizeof(unsigned char));
    
    int index;
    int W = W_I, H = H_I;

    if (imgData.size() != W*H*3) {
        printf("ERROR dimensions wahwahwah\n");
    } 
    
    /* Read into normal array (while cropping)  */
    for (index=0;index<CROPPED_AREA;index++) raw[index] = imgData[index];

    /* Use CUDA for scaling and grayscale */
    hipMemcpy(d_raw,raw,CROPPED_AREA*sizeof(unsigned char),hipMemcpyHostToDevice);
    scaleCuda<<<W_O,H_O>>>(d_raw,d_scCrop);

    /* Done with raw */
    free(raw);
    hipFree(d_raw);

    edge = (unsigned int*) malloc(AREA*sizeof(unsigned int));
    hipMalloc(&d_edge,AREA*sizeof(unsigned int));

    /* Use CUDA for edge detection */
    edgeMath<<<W_O,H_O>>>(d_scCrop,d_edge);
    hipMemcpy(edge,d_edge,AREA*sizeof(unsigned int),hipMemcpyDeviceToHost);

    /* Done with scCrop */
    free(scCrop);
    hipFree(d_scCrop);

    /* Saving to std::vector */
    for (index=0;index<AREA;index++) output.push_back((edge[index] > THRESHOLD2) ? 255 : 0);

    /* Done with edge */
    free(edge);
    hipFree(d_edge);

    return output;

}

std::vector<unsigned char> processImageDumb(std::vector<unsigned char> imgData) {
    std::vector <unsigned char> output;
    output = scaleAndCrop(imgData);
    output = toGrayscale(output);
    output = getEdges(output);
    return output;
}

std::vector<unsigned char> processImageMin(std::vector<unsigned char> imgData) {
    std::vector <unsigned char> output;
    output = scaleCropGrayscale(imgData);
    output = getEdges(output);
    return output;
}




#ifdef INPROGRESS
__global__
void findHough(unsigned char* edge, unsigned char* temp) {
    int index = blockIdx.x*blockDim.x + threadIdx.x;
    int i = index/W_O;
    int j = index%W_O;
    int x = j;
    int y = newH -1 -i;
    if (edge[index]) {
                
    }

}


void blah() {
    
  for (i=0;i<newH;i++) {
        for (j=0;j<newW;j++) {
            int x,y;
            x = j;
            y = newH - 1 - i;
            int index = newW*i + j;
            if (edge_img[index]) {
                for (theta=0;theta<180;theta++) {
                    roh = -x*sin(theta*M_PI/180) + y*cos(theta*M_PI/180);
                    roh_min = (roh_min>roh)? roh: roh_min;
                    roh_max = (roh_max<roh)? roh: roh_max;
                    voting[theta].push_back(roh);    
                }
            } 
        }
    }

    int hough_max = 0;
    int final_r, final_t;
    for (theta=0;theta<180;theta++) {
        for (roh=roh_min;roh<roh_max+1;roh++) {
            if (count(voting,theta,roh)>hough_max) {
                hough_max = count(voting,theta,roh);
                final_r = roh;
                final_t = theta;
            }
        }
    }

    printf("r %d, t %d\n", roh, theta);

    for (i=0;i<newH;i++) {
        for (j=0;j<newW;j++) {
            int x = j;
            int y = newH-1-i;
            if (abs(x*sin(final_t*M_PI/180) - y*cos(final_t*M_PI/180) + final_r) < 1) edge.data.push_back(255);
            else edge.data.push_back(0);
        }
    }


}

#endif
